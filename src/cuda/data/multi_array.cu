#include "hip/hip_runtime.h"
#include "algorithms/interpolation.h"
#include "cuda/cuda_utility.h"
#include "data/multi_array.h"
#include "data/multi_array_impl.hpp"
#include <algorithm>
#include <stdexcept>

namespace Coffee {

namespace Kernels {

template <typename T>
__global__ void
assign_single_value(T* data, size_t size, T value) {
  for (size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size;
       i += blockDim.x * gridDim.x) {
    data[i] = value;
  }
}

template <typename T>
__global__ void
downsample_average(T* orig_data, float* dst_data, Extent orig_ext,
                   Extent dst_ext, Index offset, Stagger st, int d) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  if (i < dst_ext.x && j < dst_ext.y && k < dst_ext.z) {
    size_t orig_idx = i * d + offset.x +
                      (j * d + offset.y) * orig_ext.x +
                      (k * d + offset.z) * orig_ext.x * orig_ext.y;
    size_t dst_idx = i + j * dst_ext.x + k * dst_ext.x * dst_ext.y;

    for (int kk = 0; kk < min(d, orig_ext.z); kk++) {
      for (int jj = 0; jj < min(d, orig_ext.y); jj++) {
        for (int ii = 0; ii < d; ii++) {
          dst_data[dst_idx] += interpolate(
              orig_data,
              orig_idx + ii + (jj + kk * orig_ext.y) * orig_ext.x, st,
              Stagger(0b111), orig_ext.x, orig_ext.y);
        }
      }
    }
    if (orig_ext.z > d) dst_data[dst_idx] /= d;
    if (orig_ext.y > d) dst_data[dst_idx] /= d;
    if (orig_ext.x > d) dst_data[dst_idx] /= d;
    // dst_data[dst_idx] = orig_data[orig_idx];
  }
}

template <typename T>
__global__ void
downsample(T* orig_data, float* dst_data, Extent orig_ext,
           Extent dst_ext, Index offset, Stagger st, int d) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  if (i < dst_ext.x && j < dst_ext.y && k < dst_ext.z) {
    size_t orig_idx = i * d + offset.x +
                      (j * d + offset.y) * orig_ext.x +
                      (k * d + offset.z) * orig_ext.x * orig_ext.y;
    size_t dst_idx = i + j * dst_ext.x + k * dst_ext.x * dst_ext.y;

    dst_data[dst_idx] =
        interpolate(orig_data, orig_idx, st, Stagger(0b111), orig_ext.x,
                    orig_ext.y);

    // dst_data[dst_idx] = orig_data[orig_idx];
  }
}

template <typename T>
__global__ void
downsample2d(T* orig_data, float* dst_data, Extent orig_ext,
             Extent dst_ext, Index offset, Stagger st, int d) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if (i < dst_ext.x && j < dst_ext.y) {
    size_t orig_idx =
        i * d + offset.x + (j * d + offset.y) * orig_ext.x;
    size_t dst_idx = i + j * dst_ext.x;

    dst_data[dst_idx] =
        interpolate2d(orig_data, orig_idx, st, Stagger(0b111), orig_ext.x);

    // dst_data[dst_idx] = orig_data[orig_idx];
  }
}

}  // namespace Kernels

template <typename T>
void
multi_array<T>::alloc_mem(size_t size) {
  m_data_h = new T[size];

  CudaSafeCall(hipMalloc(&m_data_d, sizeof(T) * size));
}

template <typename T>
void
multi_array<T>::free_mem() {
  if (m_data_h != nullptr) {
    delete[] m_data_h;
    m_data_h = nullptr;
  }
  if (m_data_d != nullptr) {
    CudaSafeCall(hipFree(m_data_d));
    m_data_d = nullptr;
  }
}

template <typename T>
void
multi_array<T>::copy_from(const self_type& other) {
  if (m_size != other.m_size) {
    throw std::range_error(
        "Trying to copy from a multi_array of different size!");
  }
  // memcpy(m_data_h, other.m_data_h, m_size * sizeof(T));
  CudaSafeCall(hipMemcpy(m_data_d, other.m_data_d, m_size * sizeof(T),
                          hipMemcpyDeviceToDevice));
}

template <typename T>
void
multi_array<T>::assign_dev(const T& value) {
  Kernels::assign_single_value<<<256, 512>>>(m_data_d, m_size, value);
  CudaCheckError();
}

template <typename T>
void
multi_array<T>::sync_to_host() {
  CudaSafeCall(hipMemcpy(m_data_h, m_data_d, m_size * sizeof(T),
                          hipMemcpyDeviceToHost));
}

template <typename T>
void
multi_array<T>::sync_to_device() {
  CudaSafeCall(hipMemcpy(m_data_d, m_data_h, m_size * sizeof(T),
                          hipMemcpyHostToDevice));
}

template <typename T>
void
multi_array<T>::downsample(int d, multi_array<float>& array,
                           Index offset, Stagger stagger,
                           float* h_ptr) {
  auto& ext = array.extent();
  if (ext.y == 1 && ext.z == 1) {
    // Use 1D version which we did not implement
  } else if (ext.z == 1) {  // Use 2D version
    dim3 blockSize(32, 32);
    dim3 gridSize((ext.x + blockSize.x - 1) / blockSize.x,
                  (ext.y + blockSize.y - 1) / blockSize.y);
    Kernels::downsample2d<<<gridSize, blockSize>>>(
        m_data_d, array.dev_ptr(), m_extent, array.extent(), offset,
        stagger, d);
    CudaCheckError();
  } else {
    dim3 blockSize(32, 8, 4);
    dim3 gridSize((ext.x + blockSize.x - 1) / blockSize.x,
                  (ext.y + blockSize.y - 1) / blockSize.y,
                  (ext.z + blockSize.z - 1) / blockSize.z);
    Kernels::downsample<<<gridSize, blockSize>>>(
        m_data_d, array.dev_ptr(), m_extent, array.extent(), offset,
        stagger, d);
    CudaCheckError();
  }
  // CudaSafeCall(hipMemcpy(h_ptr, array.m_data_d,
  //                         array.size() * sizeof(T),
  //                         hipMemcpyDeviceToHost));
  array.sync_to_host();
}

/////////////////////////////////////////////////////////////////
// Explicitly instantiate the classes we will use
/////////////////////////////////////////////////////////////////
template class multi_array<long long>;
template class multi_array<long>;
template class multi_array<int>;
template class multi_array<short>;
template class multi_array<char>;
template class multi_array<unsigned int>;
template class multi_array<unsigned long>;
template class multi_array<unsigned long long>;
template class multi_array<float>;
template class multi_array<double>;
template class multi_array<long double>;

}  // namespace Coffee
