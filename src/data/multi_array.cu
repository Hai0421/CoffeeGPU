#include "multi_array.h"

namespace Coffee {

template <typename T>
multi_array<T>::multi_array()
    : m_data_h(nullptr),
      m_data_d(nullptr),
      m_extent(0, 1, 1),
      m_size(0) {}

template <typename T>
multi_array<T>::multi_array(int width, int height, int depth)
    : m_extent(width, height, depth) {
  m_size = width * height * depth;

  alloc_mem(m_size);
}

template <typename T>
multi_array<T>::multi_array(const Extent& extent)
    : multi_array(extent.width(), extent.height(), extent.depth()) {}

template <typename T>
multi_array<T>::multi_array(const self_type& other)
    : multi_array(other.m_extent) {}

template <typename T>
multi_array<T>::multi_array(self_type&& other) {
  m_extent = other.m_extent;
  m_size = other.m_size;
  m_data_h = other.m_data_h;
  m_data_d = other.m_data_d;

  other.m_data_h = nullptr;
  other.m_data_d = nullptr;
}

template <typename T>
multi_array<T>::~multi_array() {
  free_mem();
}

template <typename T>
void
multi_array<T>::alloc_mem(size_t size) {
  m_data_h = new T[size];

  CudaSafeCall(hipMalloc(&m_data_d, sizeof(T) * size));
}

template <typename T>
void
multi_array<T>::free_mem() {
  if (m_data_h != nullptr) {
    delete[] m_data_h;
    m_data_h = nullptr;
  }
  if (m_data_d != nullptr) {
    CudaSafeCall(hipFree(m_data_d));
    m_data_d = nullptr;
  }
}

}  // namespace Coffee
