#include "hip/hip_runtime.h"
#include "algorithms/interpolation.h"
#include "cuda/cuda_utility.h"
#include "multi_array.h"
#include <algorithm>
#include <stdexcept>

namespace Coffee {

namespace Kernels {

template <typename T>
__global__ void
assign_single_value(T* data, size_t size, T value) {
  for (size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size;
       i += blockDim.x * gridDim.x) {
    data[i] = value;
  }
}

template <typename T>
__global__ void
downsample(T* orig_data, float* dst_data, Extent orig_ext,
           Extent dst_ext, Index offset, Stagger st, int d) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  if (i < dst_ext.x && j < dst_ext.y && k < dst_ext.z) {
    size_t orig_idx = i * d + offset.x +
                      (j * d + offset.y) * orig_ext.x +
                      (k * d + offset.z) * orig_ext.x * orig_ext.y;
    size_t dst_idx = i + j * dst_ext.x + k * dst_ext.x * dst_ext.y;

    for (int kk = 0; kk < min(d, orig_ext.z); kk++) {
      for (int jj = 0; jj < min(d, orig_ext.y); jj++) {
        for (int ii = 0; ii < d; ii++) {
          dst_data[dst_idx] += interpolate(
              orig_data,
              orig_idx + ii + (jj + kk * orig_ext.y) * orig_ext.x, st,
              Stagger(0b111), orig_ext.x, orig_ext.y);
        }
      }
    }
    if (orig_ext.z > d) dst_data[dst_idx] /= d;
    if (orig_ext.y > d) dst_data[dst_idx] /= d;
    if (orig_ext.x > d) dst_data[dst_idx] /= d;
    // dst_data[dst_idx] = orig_data[orig_idx];
  }
}

}  // namespace Kernels

template <typename T>
multi_array<T>::multi_array()
    : m_data_h(nullptr),
      m_data_d(nullptr),
      m_extent(0, 1, 1),
      m_size(0) {}

template <typename T>
multi_array<T>::multi_array(int width, int height, int depth)
    : m_extent(width, height, depth) {
  m_size = width * height * depth;

  alloc_mem(m_size);
  assign_dev(0);
}

template <typename T>
multi_array<T>::multi_array(const Extent& extent)
    : multi_array(extent.width(), extent.height(), extent.depth()) {}

template <typename T>
multi_array<T>::multi_array(const self_type& other)
    : multi_array(other.m_extent) {
  copy_from(other);
  sync_to_host();
}

template <typename T>
multi_array<T>::multi_array(self_type&& other) {
  m_extent = other.m_extent;
  m_size = other.m_size;
  m_data_h = other.m_data_h;
  m_data_d = other.m_data_d;

  other.m_data_h = nullptr;
  other.m_data_d = nullptr;
}

template <typename T>
multi_array<T>::~multi_array() {
  free_mem();
}

template <typename T>
multi_array<T>&
multi_array<T>::operator=(const self_type& other) {
  free_mem();
  m_size = other.m_size;
  m_extent = other.m_extent;

  alloc_mem(other.m_size);
  copy_from(other);
  sync_to_host();
  return *this;
}

template <typename T>
multi_array<T>&
multi_array<T>::operator=(self_type&& other) {
  m_data_h = other.m_data_h;
  m_data_d = other.m_data_d;

  other.m_data_h = nullptr;
  other.m_data_d = nullptr;

  m_extent = other.m_extent;
  m_size = other.m_size;

  return *this;
}

template <typename T>
void
multi_array<T>::alloc_mem(size_t size) {
  m_data_h = new T[size];

  CudaSafeCall(hipMalloc(&m_data_d, sizeof(T) * size));
}

template <typename T>
void
multi_array<T>::free_mem() {
  if (m_data_h != nullptr) {
    delete[] m_data_h;
    m_data_h = nullptr;
  }
  if (m_data_d != nullptr) {
    CudaSafeCall(hipFree(m_data_d));
    m_data_d = nullptr;
  }
}

template <typename T>
const T&
multi_array<T>::operator()(int x, int y, int z) const {
  size_t idx = x + (y + z * m_extent.height()) * m_extent.width();
  return m_data_h[idx];
}

template <typename T>
T&
multi_array<T>::operator()(int x, int y, int z) {
  size_t idx = x + (y + z * m_extent.height()) * m_extent.width();
  return m_data_h[idx];
}

template <typename T>
const T&
multi_array<T>::operator()(const Index& index) const {
  return operator()(index.x, index.y, index.z);
}

template <typename T>
T&
multi_array<T>::operator()(const Index& index) {
  return operator()(index.x, index.y, index.z);
}

template <typename T>
const T& multi_array<T>::operator[](size_t n) const {
  return m_data_h[n];
}

template <typename T>
T& multi_array<T>::operator[](size_t n) {
  return m_data_h[n];
}

template <typename T>
void
multi_array<T>::copy_from(const self_type& other) {
  if (m_size != other.m_size) {
    throw std::range_error(
        "Trying to copy from a multi_array of different size!");
  }
  // memcpy(m_data_h, other.m_data_h, m_size * sizeof(T));
  CudaSafeCall(hipMemcpy(m_data_d, other.m_data_d, m_size * sizeof(T),
                          hipMemcpyDeviceToDevice));
}

template <typename T>
void
multi_array<T>::assign(const T& value) {
  std::fill_n(m_data_h, m_size, value);
}

template <typename T>
void
multi_array<T>::assign_dev(const T& value) {
  Kernels::assign_single_value<<<256, 512>>>(m_data_d, m_size, value);
  CudaCheckError();
}

template <typename T>
void
multi_array<T>::resize(int width, int height, int depth) {
  size_t size = width * height * depth;
  m_extent = Extent(width, height, depth);
  m_size = size;

  // Do nothing if the sizes already match
  if (m_size == size) {
    return;
  }
  free_mem();
  alloc_mem(size);
}

template <typename T>
void
multi_array<T>::resize(Extent extent) {
  resize(extent.width(), extent.height(), extent.depth());
}

template <typename T>
void
multi_array<T>::sync_to_host() {
  CudaSafeCall(hipMemcpy(m_data_h, m_data_d, m_size * sizeof(T),
                          hipMemcpyDeviceToHost));
}

template <typename T>
void
multi_array<T>::sync_to_device() {
  CudaSafeCall(hipMemcpy(m_data_d, m_data_h, m_size * sizeof(T),
                          hipMemcpyHostToDevice));
}

template <typename T>
void
multi_array<T>::downsample(int d, multi_array<float>& array,
                           Index offset, Stagger stagger,
                           float* h_ptr) {
  auto& ext = array.extent();
  dim3 blockSize(32, 8, 4);
  dim3 gridSize((ext.x + blockSize.x - 1) / blockSize.x,
                (ext.y + blockSize.y - 1) / blockSize.y,
                (ext.z + blockSize.z - 1) / blockSize.z);
  Kernels::downsample<<<gridSize, blockSize>>>(
      m_data_d, array.dev_ptr(), m_extent, array.extent(), offset,
      stagger, d);
  CudaCheckError();

  // CudaSafeCall(hipMemcpy(h_ptr, array.m_data_d,
  //                         array.size() * sizeof(T),
  //                         hipMemcpyDeviceToHost));
  array.sync_to_host();
}

/////////////////////////////////////////////////////////////////
// Explicitly instantiate the classes we will use
/////////////////////////////////////////////////////////////////
template class multi_array<long long>;
template class multi_array<long>;
template class multi_array<int>;
template class multi_array<short>;
template class multi_array<char>;
template class multi_array<unsigned int>;
template class multi_array<unsigned long>;
template class multi_array<unsigned long long>;
template class multi_array<float>;
template class multi_array<double>;
template class multi_array<long double>;

}  // namespace Coffee
