#include "hip/hip_runtime.h"
#include "cuda/cuda_utility.h"
#include "multi_array.h"
#include <algorithm>
#include <stdexcept>

namespace Coffee {

namespace Kernels {

template <typename T>
__global__ void
assign_single_value(T* data, size_t size, T value) {
  for (size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size;
       i += blockDim.x * gridDim.x) {
    data[i] = value;
  }
}

}  // namespace Kernels

template <typename T>
multi_array<T>::multi_array()
    : m_data_h(nullptr),
      m_data_d(nullptr),
      m_extent(0, 1, 1),
      m_size(0) {}

template <typename T>
multi_array<T>::multi_array(int width, int height, int depth)
    : m_extent(width, height, depth) {
  m_size = width * height * depth;

  alloc_mem(m_size);
  assign_dev(0);
}

template <typename T>
multi_array<T>::multi_array(const Extent& extent)
    : multi_array(extent.width(), extent.height(), extent.depth()) {}

template <typename T>
multi_array<T>::multi_array(const self_type& other)
    : multi_array(other.m_extent) {
  copy_from(other);
  sync_to_host();
}

template <typename T>
multi_array<T>::multi_array(self_type&& other) {
  m_extent = other.m_extent;
  m_size = other.m_size;
  m_data_h = other.m_data_h;
  m_data_d = other.m_data_d;

  other.m_data_h = nullptr;
  other.m_data_d = nullptr;
}

template <typename T>
multi_array<T>::~multi_array() {
  free_mem();
}

template <typename T>
multi_array<T>&
multi_array<T>::operator=(const self_type& other) {
  free_mem();
  m_size = other.m_size;
  m_extent = other.m_extent;

  alloc_mem(other.m_size);
  copy_from(other);
  sync_to_host();
  return *this;
}

template <typename T>
multi_array<T>&
multi_array<T>::operator=(self_type&& other) {
  m_data_h = other.m_data_h;
  m_data_d = other.m_data_d;

  other.m_data_h = nullptr;
  other.m_data_d = nullptr;

  m_extent = other.m_extent;
  m_size = other.m_size;

  return *this;
}

template <typename T>
void
multi_array<T>::alloc_mem(size_t size) {
  m_data_h = new T[size];

  CudaSafeCall(hipMalloc(&m_data_d, sizeof(T) * size));
}

template <typename T>
void
multi_array<T>::free_mem() {
  if (m_data_h != nullptr) {
    delete[] m_data_h;
    m_data_h = nullptr;
  }
  if (m_data_d != nullptr) {
    CudaSafeCall(hipFree(m_data_d));
    m_data_d = nullptr;
  }
}

template <typename T>
const T&
multi_array<T>::operator()(int x, int y, int z) const {
  size_t idx = x + (y + z * m_extent.height()) * m_extent.width();
  return m_data_h[idx];
}

template <typename T>
T&
multi_array<T>::operator()(int x, int y, int z) {
  size_t idx = x + (y + z * m_extent.height()) * m_extent.width();
  return m_data_h[idx];
}

template <typename T>
const T&
multi_array<T>::operator()(const Index& index) const {
  return operator()(index.x, index.y, index.z);
}

template <typename T>
T&
multi_array<T>::operator()(const Index& index) {
  return operator()(index.x, index.y, index.z);
}

template <typename T>
const T& multi_array<T>::operator[](size_t n) const {
  return m_data_h[n];
}

template <typename T>
T& multi_array<T>::operator[](size_t n) {
  return m_data_h[n];
}

template <typename T>
void
multi_array<T>::copy_from(const self_type& other) {
  if (m_size != other.m_size) {
    throw std::range_error(
        "Trying to copy from a multi_array of different size!");
  }
  // memcpy(m_data_h, other.m_data_h, m_size * sizeof(T));
  CudaSafeCall(hipMemcpy(m_data_d, other.m_data_d, m_size * sizeof(T),
                          hipMemcpyDeviceToDevice));
}

template <typename T>
void
multi_array<T>::assign(const T& value) {
  std::fill_n(m_data_h, m_size, value);
}

template <typename T>
void
multi_array<T>::assign_dev(const T& value) {
  Kernels::assign_single_value<<<256, 512>>>(m_data_d, m_size, value);
  CudaCheckError();
}

template <typename T>
void
multi_array<T>::resize(int width, int height, int depth) {
  size_t size = width * height * depth;
  m_extent = Extent(width, height, depth);
  m_size = size;

  // Do nothing if the sizes already match
  if (m_size == size) {
    return;
  }
  free_mem();
  alloc_mem(size);
}

template <typename T>
void
multi_array<T>::resize(Extent extent) {
  resize(extent.width(), extent.height(), extent.depth());
}

template <typename T>
void
multi_array<T>::sync_to_host() {
  CudaSafeCall(hipMemcpy(m_data_h, m_data_d, m_size * sizeof(T),
                          hipMemcpyDeviceToHost));
}

template <typename T>
void
multi_array<T>::sync_to_device() {
  CudaSafeCall(hipMemcpy(m_data_d, m_data_h, m_size * sizeof(T),
                          hipMemcpyHostToDevice));
}

/////////////////////////////////////////////////////////////////
// Explicitly instantiate the classes we will use
/////////////////////////////////////////////////////////////////
template class multi_array<long long>;
template class multi_array<long>;
template class multi_array<int>;
template class multi_array<short>;
template class multi_array<char>;
template class multi_array<unsigned int>;
template class multi_array<unsigned long>;
template class multi_array<unsigned long long>;
template class multi_array<float>;
template class multi_array<double>;
template class multi_array<long double>;

}  // namespace Coffee
