#include "multi_array.h"

namespace Coffee {

template <typename T>
multi_array<T>::multi_array()
    : m_data_h(nullptr),
      m_data_d(nullptr),
      m_extent(0, 1, 1),
      m_size(0) {}

template <typename T>
multi_array<T>::multi_array(int width, int height, int depth)
    : m_extent(width, height, depth) {
  m_size = width * height * depth;

  alloc_mem(m_size);
}

template <typename T>
multi_array<T>::multi_array(const Extent& extent)
    : multi_array(extent.width(), extent.height(), extent.depth()) {}

template <typename T>
multi_array<T>::multi_array(const self_type& other)
    : multi_array(other.m_extent) {}

template <typename T>
multi_array<T>::multi_array(self_type&& other) {
  m_extent = other.m_extent;
  m_size = other.m_size;
  m_data_h = other.m_data_h;
  m_data_d = other.m_data_d;

  other.m_data_h = nullptr;
  other.m_data_d = nullptr;
}

template <typename T>
multi_array<T>::~multi_array() {
  free_mem();
}

template <typename T>
void
multi_array<T>::alloc_mem(size_t size) {
  m_data_h = new T[size];

  CudaSafeCall(hipMalloc(&m_data_d, sizeof(T) * size));
}

template <typename T>
void
multi_array<T>::free_mem() {
  if (m_data_h != nullptr) {
    delete[] m_data_h;
    m_data_h = nullptr;
  }
  if (m_data_d != nullptr) {
    CudaSafeCall(hipFree(m_data_d));
    m_data_d = nullptr;
  }
}

template <typename T>
const T& multi_array<T>::operator()(int x, int y, int z) const {
  size_t idx = x + (y + z * m_extent.height()) * m_extent.width();
  return m_data_h[idx];
}

template <typename T>
T& multi_array<T>::operator()(int x, int y, int z) {
  size_t idx = x + (y + z * m_extent.height()) * m_extent.width();
  return m_data_h[idx];
}

template <typename T>
const T& multi_array<T>::operator()(const Index& index) const {
  size_t idx = index.linear_index(m_extent);
  return m_data_h[idx];
}

template <typename T>
T& multi_array<T>::operator()(const Index& index) {
  size_t idx = index.linear_index(m_extent);
  return m_data_h[idx];
}

}  // namespace Coffee
